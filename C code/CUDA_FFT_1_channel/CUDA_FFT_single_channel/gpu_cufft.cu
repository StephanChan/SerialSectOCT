#include "hip/hip_runtime.h"
// Serial_code.cpp : This file contains the 'main' function. Program execution begins and ends there.
//



#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <cstdio>
#include <cstdlib>
//#include <time.h>
#include <math.h>
//#include "cuPrintf.cu"
//#include "cuPrintf.cuh"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
//#define NUM_STREAMS 4
//#define NUM_BLINES 40
//#define NBLINES 1100
//#define NALINES 1250
//#define SAMPLES 1440
//#define DEPTH   100
#define SWAP(a,b) tempr=(a); (a)=(b); (b)=tempr
#define data_t float
#define PI 3.14159265359
#define sample_f 1000
#define signal_f 100
#define PRINT_TIME 1

extern "C" __declspec(dllexport) void single_channel_process(uint16_t *h_signal, data_t *h_processed_signal, int z0, int DEPTH, int a, int b, int NALINES, int NBLINES, int SAMPLES);
int main(int argc, char **argv) {
    uint16_t *h_signal;
    data_t *h_processed_signal;
    FILE *fp;
    long long int i;
	
	int NALINES = 1250;
	int NBLINES = 1100;
	int SAMPLES = 1440;
	int DEPTH = 100;

    long long int length_raw = (long long int)SAMPLES * (long long int)NALINES * (long long int)NBLINES * (long long int)1;   //FIXME: can not initialize a number this big
    long long int length_data = (long long int)DEPTH * (long long int)1 * (long long int)NALINES * (long long int)NBLINES;
    //printf("size of h_signal is %lld, %lld\n", (long long int)length_raw, (long long int)length_raw*sizeof(uint16_t));
    h_signal = (uint16_t *)malloc(sizeof(uint16_t) * (long long int)length_raw);   //1440*1100*1250*2
	//h_FFT_buffer = (data_t *)malloc(sizeof(data_t)*NALINES * 4096 * 20);
    h_processed_signal = (data_t *)malloc(sizeof(data_t) * (long long int)length_data);  //600*1100*1250
	
    // init raw data
    /*for (i = 0; i < length_raw; i++) {
      h_signal[i] = (uint16_t)((sin(2*PI*signal_f*i/sample_f)+1)/2*65535);
    }
	*/
	fp = fopen("C:\\Users\\BOAS-USER\\Documents\\Data\\200121 layered glass sample\\spectral data\\1-1-1440-1250-1100-A0.dat","rb");
	if (fp != NULL) {
		fread(h_signal, sizeof(uint16_t), length_raw, fp);
		fclose(fp);
		printf("read file success\n");
	}
	else
		printf("open file failed\n");
	//printf("%d %d\n", h_signal[0], h_signal[100]);
	/*fp = fopen("h_signal.txt", "w");
	if (fp != NULL) {
		for (i = 0; i < NALINES * SAMPLES * 40; i++)
			fprintf(fp, "%d, %d\n", i, h_signal[i]);
		//for (i = 0; i < NALINES * SAMPLES * 40; i++)
		//	fprintf(fp, "%d, %d\n", i, h_signal[i]);
		fclose(fp);
		printf("write to file success\n");
	}
	else
		printf("open file failed\n");*/
	//for (i = 0; i < length_data; i++) {
	//	h_processed_signal[i] =-1;
	//}

    //fopen_s(&fp, "", "rb");
    //fread(h_signal, 2, length_raw, fp);
    //fclose(fp);
    //printf("finished initializing data, size of processed data: %lld\n", (long long int)length_data* sizeof(data_t));
    printf("start processing...\n");
	single_channel_process(h_signal, h_processed_signal, 0,DEPTH,0,0,NALINES, NBLINES, SAMPLES);
	printf("finished processing...\n");
	//printf("finished processing data, size of processed data: %lld\n", (long long int)length(h_processed_signal));
    /*fp=fopen("data.dat", "w");
	if (fp != NULL) {
		fwrite(h_processed_signal, sizeof(data_t), length_data, fp);
		fclose(fp);
		printf("write to file success\n");
	}
	else
		printf("open file failed\n");
		*/
	/*
	fp = fopen("GPU_h_FFT_buffer.txt", "w");
	if (fp != NULL) {
		for(i= 0;i<NALINES*4096*10;i++)
		   fprintf(fp,"%d, %f\n",i, h_FFT_buffer[i]);
		fclose(fp);
		printf("write to file success\n");
	}
	else
		printf("open file failed\n");
		*/
	/*
	fp = fopen("GPU_h_processed.txt", "w");
	if (fp != NULL) {
		for (i = 0; i < NALINES * DEPTH * NBLINES; i++)
			fprintf(fp, "%d, %.10f\n", i, h_processed_signal[i]);
		fclose(fp);
		printf("write to file success\n");
	}
	else
		printf("open file failed\n");
		*/
return 0;
}


void single_channel_process(uint16_t *h_signal, data_t *h_processed_signal, int z0, int DEPTH, int a, int b, int NALINES, int NBLINES, int SAMPLES) {
    void my_cufft(uint16_t *h_signalA, data_t *dispersion, data_t *hann, data_t *h_processed_signal, int z0, int DEPTH, int numBline, int NALINES);
	//hipEvent_t start, stop;
    //float elapsed_gpu;
    long long int i;
	int size1, size2, size3;
	//change size1, size2, size3 for difference NBLINES
	size1 = 400;
	size2 = 400;
	size3 = 300;
    data_t *dispersion, *d_dispersion, *hann, *d_hann;
    data_t f, arg;
    data_t k0,k1,dk,kc;
    data_t   *d_processed;
    uint16_t *d_dataA;
    dispersion = (data_t *) malloc(sizeof(data_t) * SAMPLES * 2);
    //init dispersion compensation array
	k0 = 2 * PI / 1363 ;
	k1 = 2 * PI / 1227 ;
	kc = 2 * PI / 1295 ;
	dk = (k1 - k0) / (SAMPLES-1);
    a = data_t(a * pow(10, 4));
    b = data_t(b * pow(10, 6));

    for (i = 0; i < SAMPLES; i ++) {
        f = 3 * (k0 - kc + (i) * dk);
        arg = a * pow(f, 2) + b * pow(f, 3);
        dispersion[2*i] = (data_t) cos(arg);
        dispersion[2*i + 1] = (data_t) sin(arg);
		//printf("%.10f, %.10f\n", dispersion[2*i],dispersion[2*i+1]);
        //dispersion[2*i] = 1;
        //dispersion[2*i+1] = 0;
    }

	//init hann window array
	hann= (data_t *)malloc(sizeof(data_t) * SAMPLES);
	for (i = 0; i < SAMPLES; i++) {
		hann[i] = 0.5*(1 - cos(2 * PI*i / (SAMPLES-1)));
		//printf("%.10f\n", hann[i]);
	}

//#if PRINT_TIME
    // Create the cuda events
//    hipEventCreate(&start);
//    hipEventCreate(&stop);
   /* hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_malloc);
    hipEventCreate(&stop_malloc);
    hipEventCreate(&start_mempyHD);
    hipEventCreate(&stop_mempyHD);
    hipEventCreate(&start_mempyDH);
    hipEventCreate(&stop_mempyDH);*/
    // Record event on the default stream
//    hipEventRecord(start, 0);
//#endif
    //hipEventRecord(start_malloc, 0);
	checkCudaErrors(hipMalloc((void **) &d_dataA, (long long int)NALINES * (long long int)size1 * (long long int)sizeof(uint16_t) * (long long int)SAMPLES*1));
	checkCudaErrors(hipMalloc((void **) &d_processed, (long long int)NALINES * size1 * sizeof(data_t) * 1 * DEPTH));
	checkCudaErrors(hipMalloc((void **) &d_dispersion, sizeof(data_t) * SAMPLES * 2));
	checkCudaErrors(hipMalloc((void **)&d_hann, sizeof(data_t) * SAMPLES));
    //hipEventRecord(stop_malloc, 0);
    //hipEventSynchronize(stop_malloc);

	//hipMemset(d_processed, 0, (long long int)NALINES * size1 * sizeof(data_t) * 1 * DEPTH);
 //first part of data
	
    //hipEventRecord(start_mempyHD, 0);
	checkCudaErrors(
            hipMemcpy(d_dataA, h_signal, (long long int)NALINES * size1 * sizeof(uint16_t) * SAMPLES*1, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(d_dispersion, dispersion, sizeof(data_t) * SAMPLES * 2, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_hann, hann, sizeof(data_t) * SAMPLES, hipMemcpyHostToDevice));
   //// hipEventRecord(stop_mempyHD, 0);
   // hipEventSynchronize(stop_mempyHD);
	//printf("finished memcopy host to device first half\n");
//    cudaPrintfInit();
    // Launch the kernel
    //dim3 dimBlock(32, 32, 1);
    //dim3 dimGrid(9, 39, 1);
    //hipEventRecord(start_kernel, 0);
    //kernel_process << < dimGrid, dimBlock >> > (d_dataA, d_dataB, d_dispersion, d_processed, z0);
    my_cufft(d_dataA, d_dispersion, d_hann, d_processed, z0, DEPTH, size1, NALINES);
   // hipEventRecord(stop_kernel, 0);
   // hipEventSynchronize(stop_kernel);
//    cudaPrintfDisplay(stdout, true);
//    cudaPrintfEnd();
    // Check for errors during launch
    //CUDA_SAFE_CALL(hipPeekAtLastError());
    // Transfer the results back to the host
	//printf("finished process first half\n");
   // hipEventRecord(start_mempyDH, 0);
	checkCudaErrors(hipMemcpy(h_processed_signal, d_processed, NALINES * size1 * sizeof(data_t) * 1 * DEPTH,
                              hipMemcpyDeviceToHost));
   // hipEventRecord(stop_mempyDH, 0);
   // hipEventSynchronize(stop_mempyDH);
	//printf("finished memcopy device to host for first half\n");
	
 //second part
	//checkCudaErrors(hipMemcpy(d_dispersion, dispersion, sizeof(data_t) * 2048 * 2, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy(d_hann, hann, sizeof(data_t) * SAMPLES, hipMemcpyHostToDevice));
    checkCudaErrors(
            hipMemcpy(d_dataA, h_signal+(long long int)NALINES*size1*SAMPLES*1, (long long int)NALINES * size2 * sizeof(uint16_t) * SAMPLES*1, hipMemcpyHostToDevice));
   
    //hipEventRecord(start_kernel, 0);
    //kernel_process << < dimGrid, dimBlock >> > (d_dataA, d_dataB, d_dispersion, d_processed, z0);
    my_cufft(d_dataA, d_dispersion, d_hann, d_processed, z0, DEPTH, size2, NALINES);
	//printf("finished process second half\n");
    //hipEventRecord(stop_kernel, 0);
    //hipEventSynchronize(stop_kernel);
    //cudaPrintfDisplay(stdout, true);
   // cudaPrintfEnd();
    // Check for errors during launch
    //CUDA_SAFE_CALL(hipPeekAtLastError());
    // Transfer the results back to the host
	checkCudaErrors(hipMemcpy(h_processed_signal+(long long int)NALINES *size1 * 1 * DEPTH, d_processed, (long long int)NALINES * size2 * sizeof(data_t) * 1 * DEPTH,
                              hipMemcpyDeviceToHost));
//third part
	checkCudaErrors(
		hipMemcpy(d_dataA, h_signal + (long long int)NALINES*(size1+size2)*SAMPLES *1, (long long int)NALINES * size3 * sizeof(uint16_t)  * SAMPLES*1, hipMemcpyHostToDevice));
	
	//hipEventRecord(start_kernel, 0);
	//kernel_process << < dimGrid, dimBlock >> > (d_dataA, d_dataB, d_dispersion, d_processed, z0);
	my_cufft(d_dataA, d_dispersion, d_hann, d_processed, z0, DEPTH, size3, NALINES);
	//printf("finished process third half\n");
	//hipEventRecord(stop_kernel, 0);
	//hipEventSynchronize(stop_kernel);
	//cudaPrintfDisplay(stdout, true);
   // cudaPrintfEnd();
	// Check for errors during launch
	//CUDA_SAFE_CALL(hipPeekAtLastError());
	// Transfer the results back to the host
	checkCudaErrors(hipMemcpy(h_processed_signal + (long long int)NALINES * (size1+size2) * 1 * DEPTH, d_processed, (long long int)NALINES * size3 * sizeof(data_t) * 1 * DEPTH,
		hipMemcpyDeviceToHost));
	
		
	checkCudaErrors(hipFree(d_dataA));
	checkCudaErrors(hipFree(d_processed));
	checkCudaErrors(hipFree(d_dispersion));
	checkCudaErrors(hipFree(d_hann));
	//printf("finished process\n");
	/*
#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU time: %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
	/*
    hipEventElapsedTime(&elapsed_gpu, start_kernel, stop_kernel);
    printf("\nkernel time: %f (msec)\n", elapsed_gpu*2.75);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);
	
    hipEventElapsedTime(&elapsed_gpu, start_malloc, stop_malloc);
    printf("\nmalloc time: %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start_malloc);
    hipEventDestroy(stop_malloc);

    hipEventElapsedTime(&elapsed_gpu, start_mempyHD, stop_mempyHD);
    printf("\nmemcpy Host to Device time: %f (msec)\n", elapsed_gpu*2.75);
    hipEventDestroy(start_mempyHD);
    hipEventDestroy(stop_mempyHD);

    hipEventElapsedTime(&elapsed_gpu, start_mempyDH, stop_mempyDH);
    printf("\nmemcpy Device to Host time: %f (msec)\n", elapsed_gpu*2.75);
    hipEventDestroy(start_mempyDH);
    hipEventDestroy(stop_mempyDH);
#endif
	*/
    hipDeviceReset();
}


static __global__ void copy_data(long long int length_fft_data, int part, data_t *FFT_buffer, uint16_t *d_signal, data_t *hann) {
	//copy data to compex FFT buffer
	const int blockID = blockIdx.x + blockIdx.y*gridDim.x;
	const int threadID = blockID * (blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
	const int numThreads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;
	//cuPrintf("blockID: %d, threadID: %d, numTHreads: %d, datanum:%d\n", blockID, threadID, numThreads, length_fft_data);
	long long int i;
	for (i = threadID; i < length_fft_data; i += numThreads) {
		long long int k = (i % 1440) * 2 + long long int(i / 1440) * 4096;
	    FFT_buffer[k] = (d_signal[i + (long long int)part*length_fft_data]*0.8/65535-0.4)*hann[i % 1440];
		//cuPrintf("FFT_buffer %d is %f\n", i + (long long int)part*length_fft_data, FFT_buffer[k]);
	}
	
}

static __global__ void my_hilbert(data_t *FFT_buffer, long long int length_fft) {
	const int blockID = blockIdx.x + blockIdx.y*gridDim.x;
	const int threadID = blockID * (blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
	const int numThreads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;

	long long int i;
	for (i = threadID; i < long long int(length_fft / 2); i += numThreads) {
		//new_data[i % 1440*2 + i / 1440 * 4096] = old_data[i]
		long long int k =( i % 2048) + long long int(i / 2048) * 4096;
		if (i % 2048 > 1) {
			FFT_buffer[k] = 2 * FFT_buffer[k] / 2048;
			FFT_buffer[k + 2048] = 0;
			//cuPrintf("FFT_buffer is %f, %f\n", FFT_buffer[k], FFT_buffer[k + 2048]);
		}
		else {
			FFT_buffer[k] = FFT_buffer[k] / 2048;
			FFT_buffer[k + 2048] = 0;
		}
	}
}

static __global__ void my_dispersion(data_t *FFT_buffer, data_t *dispersion, long long int length_fft_data) {
	const int blockID = blockIdx.x + blockIdx.y*gridDim.x;
	const int threadID = blockID * (blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
	const int numThreads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;
	long long int i;
	for (i = threadID; i < length_fft_data; i += numThreads) {
		long long int k = (i % 1440) * 2 + long long int(i / 1440) * 4096;
		int d = (i % 1440) * 2;
	//dispersion compensation
		data_t a = FFT_buffer[k];
		data_t b = FFT_buffer[k+1];
		FFT_buffer[k] = (a * dispersion[d] - b * dispersion[d + 1]);
		FFT_buffer[k + 1] = (a * dispersion[d + 1] + b * dispersion[d]);
		//cuPrintf("FFT_buffer %d is %.10f, %.10f\n", k, FFT_buffer[k], FFT_buffer[k+1]);
	}
}
/*
static __global__ void calc(data_t *FFT_buffer, int num_Blines_per_FFT, int part, data_t *d_processed, int z0, int DEPTH, int NALINES) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	long long int i;
	long long range = NALINES * DEPTH *num_Blines_per_FFT;
	long long int step = part * num_Blines_per_FFT*NALINES*DEPTH * 3;
	for (i = threadID; i < range; i += numThreads) {
		long long int k = i % DEPTH *3  + i *3;
		long long int d = i % (DEPTH*NALINES)/DEPTH*4096 + i / DEPTH /NALINES*NALINES*1440*2 + (i % (DEPTH*NALINES) % DEPTH+z0)*3;
			data_t a = (data_t)(pow(FFT_buffer[d], 2) + pow(FFT_buffer[d + 1], 2));
			data_t p = (data_t)atan(FFT_buffer[d + 1] / FFT_buffer[d]);
			data_t a2 = (data_t)(pow(FFT_buffer[d+4096*NALINES], 2) + pow(FFT_buffer[d + 1 + 4096 * NALINES], 2));
			data_t p2 = (data_t)atan(FFT_buffer[d + 1 + 4096 * NALINES] / FFT_buffer[d + 4096 * NALINES]);

			d_processed[k + step] = a + a2;
			d_processed[k+1 + step] = (data_t)atan(sqrt(a / a2));
			d_processed[k+2 + step] = p - p2;
		
	}
}
*/
static __global__ void calc_amp(data_t *FFT_buffer, int num_Blines_per_FFT, int part, data_t *d_processed, int z0, int DEPTH, int NALINES) {
	const int blockID = blockIdx.x + blockIdx.y*gridDim.x;
	const int threadID = blockID * (blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
	const int numThreads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;

	long long int i;
	long long range = (long long int)num_Blines_per_FFT*NALINES * DEPTH;
	long long int step = (long long int)part * num_Blines_per_FFT*NALINES*DEPTH * 1;
	for (i = threadID; i < range; i += numThreads) {
		long long int k = i;   //
		long long int d = long long int(i / DEPTH) *4096 + (i % DEPTH + z0) * 2;
		data_t a = (data_t)sqrt(pow(FFT_buffer[d]/1440, 2) + pow(FFT_buffer[d + 1]/1440, 2));
		//data_t p = (data_t)atan(FFT_buffer[d + 1] / FFT_buffer[d]);
		//data_t a2 = (data_t)(pow(FFT_buffer[d + 4096 * NALINES], 2) + pow(FFT_buffer[d + 1 + 4096 * NALINES], 2));
		//data_t p2 = (data_t)atan(FFT_buffer[d + 1 + 4096 * NALINES] / FFT_buffer[d + 4096 * NALINES]);
		//cuPrintf("i is %d, d is %d, a is %f, FFT_buffer is%f\n", i, d, a, FFT_buffer[d]);
		d_processed[k + step] = a;
		//d_processed[k + 1 + step] = (data_t)atan(sqrt(a / a2));
		//d_processed[k + 2 + step] = p - p2;

	}
}

/*
static __global__ void printf_data(data_t *FFT_buffer, long long int length_fft) {
	const int blockID = blockIdx.x + blockIdx.y*gridDim.x;
	const int threadID = blockID * (blockDim.x*blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
	const int numThreads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;

	long long int i;
	for (i = threadID; i < 10; i += numThreads) {
		cuPrintf("i: %d, FFT_buffer:%.10f\n", i, sqrt(FFT_buffer[i+20*1250*100]));
	}
}
*/
void my_cufft(uint16_t *d_signal, data_t *dispersion, data_t *hann, data_t *d_processed, int z0, int DEPTH, int numBline, int NALINES) {
    int num_Blines_per_FFT;
    int part;
    long long int length_fft, length_fft_data;
    num_Blines_per_FFT=20;
    length_fft=(long long int)num_Blines_per_FFT*NALINES*4096;
    length_fft_data=(long long int)num_Blines_per_FFT*NALINES*1440;
	data_t *FFT_buffer;
    //hipfftHandle* plan = (hipfftHandle *)malloc(sizeof(hipfftHandle));
	hipfftHandle plan;
	hipfftPlan1d(&plan, 2048, HIPFFT_C2C, (long long int)NALINES * num_Blines_per_FFT);
	checkCudaErrors(hipMalloc((void **) &FFT_buffer, sizeof(data_t) * length_fft));
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid(20, 20, 1);
	//dim3 a(1, 1, 1);
	//dim3 b(1, 1, 1);
    

    for(part=0; part< numBline/num_Blines_per_FFT; part++){
		//printf("part= %d   ,length_fft=%lld, length_fft_data=%lld , %ld\n", part, length_fft, length_fft_data, plan);
        //copy data to compex FFT buffer
		hipMemset(FFT_buffer, 0, length_fft * (long long int)sizeof(data_t));
		copy_data << <dimBlock , dimGrid>> > (length_fft_data, part, FFT_buffer, d_signal, hann);

		//printf_data << <a, b >> > (FFT_buffer, length_fft);
		//printf_data << <a, b >> > (FFT_buffer, length_fft);
        //Hilbert transform
        checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(FFT_buffer), reinterpret_cast<hipfftComplex *>(FFT_buffer), HIPFFT_FORWARD));
        
		//printf_data << <a, b >> > (FFT_buffer, length_fft);
		my_hilbert << <dimBlock, dimGrid>> > ((data_t *)FFT_buffer, length_fft);
		//printf_data << <a, b >> > (FFT_buffer, length_fft);
		//printf("finished hilbert\n");
        checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(FFT_buffer), reinterpret_cast<hipfftComplex *>(FFT_buffer), HIPFFT_BACKWARD));
		//printf_data << <a, b >> > (FFT_buffer, length_fft);
		//printf_data << <a, b >> > (dispersion, length_fft);
        //dispersion compensation
		my_dispersion << <dimBlock, dimGrid>> > ((data_t *)FFT_buffer, dispersion, length_fft_data);
		//printf_data << <a, b >> > (FFT_buffer, length_fft);
        //to space domain
        checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(FFT_buffer), reinterpret_cast<hipfftComplex *>(FFT_buffer), HIPFFT_FORWARD));
		//printf_data << <a, b >> > (FFT_buffer, length_fft);
		calc_amp << <dimBlock, dimGrid>> > ((data_t *)FFT_buffer, num_Blines_per_FFT, part, d_processed, z0, DEPTH, NALINES);
		//checkCudaErrors(hipMemcpy(h_FFT_buffer, FFT_buffer, length_fft * sizeof(data_t),
		//	hipMemcpyDeviceToHost));
		//printf_data << <a, b >> > (d_processed, length_fft);
    }
	
	checkCudaErrors(hipFree(FFT_buffer));
	//checkCudaErrors(hipFree(plan));
}


// Run program: Ctrl + F5 or Debug > Start Without Debugging menu
// Debug program: F5 or Debug > Start Debugging menu

// Tips for Getting Started: 
//   1. Use the Solution Explorer window to add/manage files
//   2. Use the Team Explorer window to connect to source control
//   3. Use the Output window to see build output and other messages
//   4. Use the Error List window to view errors
//   5. Go to Project > Add New Item to create new code files, or Project > Add Existing Item to add existing code files to the project
//   6. In the future, to open this project again, go to File > Open > Project and select the .sln file
