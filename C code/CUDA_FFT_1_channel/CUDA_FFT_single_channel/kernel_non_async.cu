#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include<time.h>

#define NUM_STREAMS 4
#define NUM_BLINES 40
#define NBLINES 960
// Complex data type
typedef float2 Complex;

//static __device__ __host__ inline void ComplexPointwiseCalc(float *signal, float *temp_signal, int k);
static __global__ void ComplexPointwiseCalc(float *d_trimmed_signal, float *d_FFT_signal, int num_aline, int start, int depth, uint16_t *curve);
static __global__ void PadData(float *, uint16_t *, int num_aline);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(uint16_t *h_signal, float *h_trimmed_signal, uint16_t *d_signal, float *d_padded_signal, float *d_FFT_signal, float *d_trimmed_signal, hipfftHandle *plans, hipStream_t *streams, int num_aline, int start, int depth, uint16_t *curve);

extern "C" __declspec(dllexport)void multi_runtest(uint16_t *h_signal, float *h_trimmed_signal, int num_aline, int start, int depth, uint16_t *curve);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
	uint16_t *h_signal;
	clock_t start, stop;
	h_signal = (uint16_t *)malloc(sizeof(uint16_t) * 1440 * 2200 * NBLINES);
	float *h_trimmed_signal;
	h_trimmed_signal = (float *)malloc(sizeof(float) * 200 * 2200 * NBLINES);

	//for (int i = 0; i < 1440 * 2000* 700; i++) {
	//	h_signal[i] = rand();
	//}
	//printf("%f\n", (h_signal + 19 * 1440 * 2000 * 40)[0]);
	uint16_t *curve;
	curve = (uint16_t *)malloc(sizeof(uint16_t) * 1100 * NBLINES);
	memset(curve, 1, 1100 * NBLINES * sizeof(uint16_t));

	printf("start kernel function\n");
	start = clock();
	multi_runtest(h_signal, h_trimmed_signal, 1100, 50, 100, curve);

	stop = clock();
	printf("time for whole program is %f s\n", (double)(stop - start) / CLOCKS_PER_SEC);
	printf("%f\n", h_trimmed_signal[10]);
	printf("%f\n", h_trimmed_signal[12]);
	printf("%f\n", h_trimmed_signal[14]);
	printf("%f\n", h_trimmed_signal[16]);
	printf("%f\n", h_trimmed_signal[18]);
	printf("%f\n", h_trimmed_signal[20]);
	printf("%f\n", h_trimmed_signal[22]);
	printf("%f\n", h_trimmed_signal[24]);
	printf("%f\n", h_trimmed_signal[26]);
	//printf("h_padded_signal %f \n", h_padded_signal[1]);
	return 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void multi_runtest(uint16_t *h_signal, float *h_trimmed_signal, int num_aline, int start, int depth, uint16_t *curve) {
	//create streams
	hipStream_t streams[NUM_STREAMS];
	for (int i = 0; i < NUM_STREAMS; i++) { checkCudaErrors(hipStreamCreate(&streams[i])); }
	//create plan for FFT and assign each plan to different stream
	hipfftHandle* plans = (hipfftHandle*)malloc(sizeof(hipfftHandle)*NUM_STREAMS);
	for (int i = 0; i < NUM_STREAMS; i++) {
		hipfftPlan1d(&plans[i], 2048, HIPFFT_R2C, 2 * num_aline * NUM_BLINES / NUM_STREAMS);
		//hipfftSetStream(plans[i], streams[i]);
	}
	//init device memory
	uint16_t *d_signal;
	float *d_padded_signal;
	float *d_trimmed_signal;
	float *d_FFT_signal;
	uint16_t *d_curve;
	checkCudaErrors(hipMalloc((void **)&d_signal, sizeof(uint16_t) * 1440 * 2 * num_aline * NUM_BLINES));
	checkCudaErrors(hipMalloc((void **)&d_padded_signal, sizeof(float) * 2048 * 2 * num_aline * NUM_BLINES));
	checkCudaErrors(hipMalloc((void **)&d_FFT_signal, sizeof(float) * 2050 * 2 * num_aline * NUM_BLINES));
	checkCudaErrors(hipMalloc((void **)&d_trimmed_signal, sizeof(float) * 2 * depth * 2 * num_aline * NUM_BLINES));
	checkCudaErrors(hipMalloc((void **)&d_curve, sizeof(uint16_t) *NBLINES*num_aline));

	checkCudaErrors(hipMemcpy(d_curve, curve, sizeof(uint16_t) *NBLINES * num_aline, hipMemcpyHostToDevice));

	int block_size = 320;
	//divide dataset to 5 small groups of 100 blines
	for (int k = 0; k < NBLINES/ block_size; k++) {
		//intf("k is %d\n", k);
		//for async memcpy, memory needs to be registerred first
		//checkCudaErrors(hipHostRegister(h_signal + k * 1440 * 2 * num_aline * block_size, sizeof(uint16_t) * 1440 * 2 * num_aline * block_size, hipHostRegisterPortable));
		//checkCudaErrors(hipHostRegister(h_trimmed_signal + k * 2 * depth * 2 * num_aline * block_size, sizeof(float) * 2 * depth * 2 * num_aline * block_size, hipHostRegisterPortable));
		//divide each group to 2 sub group with 50 blines
		for (int j = 0; j < block_size/NUM_BLINES; j++) {
			//ntf("j is %d\n", j);
			checkCudaErrors(hipMemset(d_padded_signal, 0, NUM_BLINES * sizeof(float) * 2048 * 2 * num_aline));
			runTest(h_signal + k * 1440 * 2 * num_aline * block_size + j * 1440 * 2 * num_aline * NUM_BLINES, h_trimmed_signal + k * 2 * depth * 2 * num_aline * block_size + j * 2 * depth * 2 * num_aline * NUM_BLINES, d_signal, d_padded_signal, d_FFT_signal, d_trimmed_signal, plans, streams, num_aline, start, depth, d_curve + k * num_aline * block_size + j * num_aline * NUM_BLINES);
		}
		//checkCudaErrors(hipHostUnregister(h_signal + k * 1440 * 2 * num_aline * block_size));
		//checkCudaErrors(hipHostUnregister(h_trimmed_signal + k * 2 * depth * 2 * num_aline * block_size));
	}

	checkCudaErrors(hipFree(d_signal));
	checkCudaErrors(hipFree(d_trimmed_signal));
	checkCudaErrors(hipFree(d_padded_signal));
	for (int i = 0; i < NUM_STREAMS; i++) {
		checkCudaErrors(hipStreamDestroy(streams[i]));
	}
	hipDeviceReset();
}

void runTest(uint16_t *h_signal, float *h_trimmed_signal, uint16_t *d_signal, float *d_padded_signal, float *d_FFT_signal, float *d_trimmed_signal, hipfftHandle *plans, hipStream_t *streams, int num_aline, int start, int depth, uint16_t *curve) {
	//findCudaDevice(argc, (const char **)argv);

	//init streams
	for (int i = 0; i < NUM_STREAMS; i++) {
		//printf("i is %d      ", i);
		checkCudaErrors(hipMemcpy(d_signal + i * 1440 * 2 * num_aline * NUM_BLINES / NUM_STREAMS, h_signal + i * 1440 * 2 * num_aline * NUM_BLINES / NUM_STREAMS, 1440 * 2 * num_aline * sizeof(uint16_t) *NUM_BLINES / NUM_STREAMS, hipMemcpyHostToDevice)); //use same or different streams for memcopy and computation
		//Pad Data
		//checkCudaErrors(hipEventRecord(Start, 0));
		//dim3 threadsPerBlock(32, 32);

		PadData << <16, 1024, 0>> > (d_padded_signal + i * 2048 * 2 * num_aline * NUM_BLINES / NUM_STREAMS, d_signal + i * 1440 * 2 * num_aline * NUM_BLINES / NUM_STREAMS, num_aline);
		//checkCudaErrors(hipEventRecord(Stop, 0));
		//hipEventSynchronize(Stop);
		//checkCudaErrors(hipEventElapsedTime(&ms, Start, Stop));
		//printf("time for zero padding : %f ms\n", ms);

		// Transform signal and kernel
		//checkCudaErrors(hipEventRecord(Start, 0));
		checkCudaErrors(hipfftExecR2C(plans[0], reinterpret_cast<hipfftReal *>(d_padded_signal + i * 2048 * 2 * num_aline * NUM_BLINES / NUM_STREAMS),
			reinterpret_cast<hipfftComplex *>(d_FFT_signal + i * 2050 * 2 * num_aline * NUM_BLINES / NUM_STREAMS)));

		//checkCudaErrors(hipEventRecord(Stop, 0));
		//hipEventSynchronize(Stop);
		//checkCudaErrors(hipEventElapsedTime(&ms, Start, Stop));
		//printf("time for FFT with cuda event timer: %f ms\n", ms);


		// Multiply the coefficients together and normalize the result
		//checkCudaErrors(hipEventRecord(Start, 0));
		ComplexPointwiseCalc << <16, 1024, 0 >> > (d_trimmed_signal + i * 2 * depth * 2 * num_aline * NUM_BLINES / NUM_STREAMS, d_FFT_signal + i * 2050 * 2 * num_aline * NUM_BLINES / NUM_STREAMS, num_aline, start, depth, curve + i * num_aline * NUM_BLINES / NUM_STREAMS);
		//checkCudaErrors(hipEventRecord(Stop, 0));
		//hipEventSynchronize(Stop);
		//hipEventElapsedTime(&ms, Start, Stop);
		//printf("time for pointwise calculation: %f ms\n", ms);

		// Check if kernel execution generated and error
		//getLastCudaError("Kernel execution failed [ ComplexPointwiseCalc ]");
		checkCudaErrors(hipMemcpy(h_trimmed_signal + i * 2 * depth * 2 * num_aline * NUM_BLINES / NUM_STREAMS, d_trimmed_signal + i * 2 * depth * 2 * num_aline * NUM_BLINES / NUM_STREAMS, depth * 2 * 2 * num_aline * sizeof(float) * NUM_BLINES / NUM_STREAMS, hipMemcpyDeviceToHost));
	}
	//printf("\n");

	for (int i = 0; i < NUM_STREAMS; i++) {
		//printf("i is %d      ", i);
		checkCudaErrors(hipStreamSynchronize(streams[i]));
	}

}

// Pad data



////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

static __global__ void ComplexPointwiseCalc(float *d_trimmed_signal, float *d_FFT_signal, int num_aline, int start, int depth, uint16_t *curve) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int d = threadID; d < depth * 2 * num_aline * NUM_BLINES / NUM_STREAMS; d += numThreads) {
		int pos = d / depth * 1025 + d % depth + start*2 +curve[d / depth / 2];
		float cx = pow(d_FFT_signal[2 * pos], 2) + pow(d_FFT_signal[2 * pos + 1], 2);
		float cy = atan(d_FFT_signal[2 * pos + 1] / d_FFT_signal[2 * pos]);
		d_trimmed_signal[2 * d] = cx;
		d_trimmed_signal[2 * d + 1] = cy;
	}
}

static __global__ void PadData(float *new_data, uint16_t *old_data, int num_aline) {
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	//const int numThreads = blockDim.x * blockDim.y * gridDim.x;
	//const int threadID = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x + threadIdx.y * blockDim.x;

	for (int i = threadID; i < 2880 * num_aline * NUM_BLINES / NUM_STREAMS; i += numThreads) {
		new_data[i % 1440 + i / 1440 * 2048] = old_data[i];

	}

}